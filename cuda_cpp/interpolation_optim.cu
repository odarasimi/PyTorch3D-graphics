#include "hip/hip_runtime.h"
#include <torch/extension.h>

/* Nvidia T4 Tensor Core GPU: https://developer.nvidia.com/blog/nvidia-turing-architecture-in-depth/

  +---------------+
 /               /|
/               / |
+--------------+  +
|              | /
|              |/
+--------------+

*/

//-------------------------------------CALLED AND EXECUTED ON GPU-------------------------------------------------------------------------
/*
__device__ float doubleValue(float x)	
{	
	return x/2;	
}
*/

//-----------------------------------CALLED ON CPU, SPMD EXECUTION ON GPU-----------------------------------------------------------------
template <typename float>
__global__ void trilinear_kernel(
    //difference with Accessor is that a Packed Accessor copies size and stride data inside of its structure instead of pointing to it
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> vertex_features,
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> points,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> interpolated_feature
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n<feats.size(0) || f<feats.size(2))
        
        const scalar_t u = (points[n][0]+1)/2;
        const scalar_t v = (points[n][1]+1)/2;
        const scalar_t w = (points[n][2]+1)/2;
        
        const scalar_t a = (1-v)*(1-w);
        const scalar_t b = (1-v)*w;
        const scalar_t c = v*(1-w);
        const scalar_t d = 1-a-b-c;
        interpolated_feature[n][f] = (1-u)*(a*vertex_features[n][0][f] +
                                b*vertex_features[n][1][f] +
                                c*vertex_features[n][2][f] +
                                d*vertex_features[n][3][f]) + 
                                u*(a*vertex_features[n][4][f] +
                                b*vertex_features[n][5][f] +
                                c*vertex_features[n][6][f] +
                                d*vertex_features[n][7][f]);
}




//--------------------------------------SERIAL EXECUTION ON CPU-----------------------------------------------------------------
torch::Tensor cuda_trilinear(torch::Tensor vertex_features, torch::Tensor points){

    // const variables N & F to determine the shape of our output dimensions
    const int N = vertex_features.size(0), F = vertex_features.size(2);

    //tensor containing our 2 output dimensions
    torch::Tensor interpolated_feature = torch::zeros({N, F}, vertex_features.options()); 

    // determine thread & block shape 
    const int threads(16, 16);
    //dim3: data structure that encapsulates three unsigned integers: x, y, and z; z defaults to 1
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y-1)/threads.y) 

    // Kernel launch; AT_DISPATCH... = pytorch macro for dispatching operations based on the data type of the torch tensor 
    // packed accessor = tensor type conversion; 3 and 2 = num of dimensions
    AT_DISPATCH_FLOATING_TYPES(vertex_features.type(), "cuda_trilinear", 
    ([&] {
        trilinear_kernel<<<blocks, threads>>>(
            vertex_features.packed_accessor<float, 3, torch::RestrictPtrTraits>(),
            points.packed_accessor<float, 2, torch::RestrictPtrTraits>(),
            interpolated_feature.packed_accessor<float, 2, torch::RestrictPtrTraits>()
        );
    }));

    return interpolated_feature;
}
