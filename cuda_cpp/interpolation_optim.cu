#include <torch/extension.h>

/* Nvidia T4 Tensor Core GPU: https://developer.nvidia.com/blog/nvidia-turing-architecture-in-depth/

  +---------------+
 /               /|
/               / |
+--------------+  +
|              | /
|              |/
+--------------+

*/

torch::Tensor cuda_trilinear(torch::Tensor vertex_features, torch::Tensor point){
    const int N = vertex_features.size(0), F = vertex_features.size(2)

    torch::Tensor interpolated_feature = torch::zeros({N, F}, vertex_features.options())
    // determine thread size and shared memory usage after going through the nvidia turing arch
}